#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

#define M 128           // Number of BS antennas
#define N 32            // Number of users
#define SC  1           // Number of subcarriers
#define SYM 1          // Number of symbols
#define NCH SC*SYM      // Number of channel matrices
#define SZ  SC*SYM*N    // Total threads

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(-10*error); \
    } \
}

#define CHECK_CUBLAS(call)                                                     \
{                                                                              \
    hipblasStatus_t err;                                                        \
    if ((err = (call)) != HIPBLAS_STATUS_SUCCESS)                               \
    {                                                                          \
        fprintf(stderr, "Got CUBLAS error %d at %s:%d\n", err, __FILE__,       \
                __LINE__);                                                     \
        exit(1);                                                               \
    }                                                                          \
}

void createA(hipComplex *A)
{
    for (int i = 0; i < N*SZ; i++)
    {
        A[i].x = 0.123f;    
        A[i].y = 0.123f;     
    }
}

void createYMF(hipComplex *y)
{
    for (int i = 0; i < SZ; i++)
    {
        y[i].x = 0.123f;    
        y[i].y = 0.456f;     
    }
}

__global__ void ScaleA (hipComplex *A, float a)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int d = idx + threadIdx.x;

    if (idx < SZ)
    {
        A[d].x = a * A[d].x;
        A[d].y = a * A[d].y;
    }
}

int main (void)
{
    hipblasHandle_t handle;

    // Initialize cuBLAS library
    CHECK_CUBLAS (hipblasCreate(&handle));
    
    // Get device information
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);

    // Set up device
	CHECK (hipSetDevice(dev));

    // al, bet - scalars for cuBLAS
    hipComplex  al = {1.0f, 0.0f};
    hipComplex  bet = {1.0f, 0.0f};

    hipComplex * H;          // Chanel matrix in host
    hipComplex * HH;         // Hermitian Chanel matrix in host

    hipComplex * H_dev;      // Chanel matrix in device
    hipComplex * HH_dev;     // Hermitian Chanel matrix in device

    hipComplex * G;          // Gram matrix in device

    hipComplex * y;          // Received vector in host
    hipComplex * y_dev;      // Received vector in device

    hipComplex * yMF_H;      // Matched filter in host
    hipComplex * yMF;        // Matched filter in device

    hipComplex * A;          // Matrix A in host

    hipComplex * x_H;        // Detected vector 

    // Five vectors of storage are needed in addition to the matrix A: x, p, Ap, r, Ar
    hipComplex * x;
    hipComplex * p;
    hipComplex * e; // Ap
    hipComplex * r;
    hipComplex * m; // Ar

    // al, bet - scalars
    float alpha_divisor;

    hipComplex alpha = {0.0f, 0.0f};
    hipComplex alpha_divident;
    hipComplex beta = {0.0f, 0.0f};
    hipComplex beta_divident;
    hipComplex beta_divisor;

    // Matrix H in host
    H = (hipComplex*) malloc (NCH * M * N * sizeof(hipComplex));
    if (!H) {
        printf ("Host memory allocation failed for H");
        return EXIT_FAILURE;
    }

    // Matrix H in host
    HH = (hipComplex*)malloc (NCH * M * N * sizeof(hipComplex));
    if (!HH) {
        printf ("Host memory allocation failed for HH");
        return EXIT_FAILURE;
    }

    // Received vector y in host
    y = (hipComplex*)malloc (NCH * M * sizeof(hipComplex));
    if (!y) {
        printf ("Host memory allocation failed for y");
        return EXIT_FAILURE;
    }

    // Detected vector y in host
    yMF_H = (hipComplex*)malloc (NCH * N * sizeof(hipComplex));
    if (!yMF_H) {
        printf ("Host memory allocation failed for yMF_H");
        return EXIT_FAILURE;
    }

    // Detected vector y in host
    x_H = (hipComplex*)malloc (NCH * N * sizeof(hipComplex));
    if (!x_H) {
        printf ("Host memory allocation failed for x_H");
        return EXIT_FAILURE;
    }

    // Matrix H in host
    A = (hipComplex*)malloc (NCH * N * N * sizeof(hipComplex));
    if (!A) {
        printf ("Host memory allocation failed for A");
        return EXIT_FAILURE;
    }

    // Allocate matrix H in device
    CHECK (hipMalloc ((void**)&H_dev, NCH*M*N*sizeof(hipComplex)));

    // Allocate matrix HH in device
    CHECK (hipMalloc ((void**)&HH_dev, NCH*M*N*sizeof(hipComplex)));

    // Allocate vector y in device
    CHECK (hipMalloc ((void**)&y_dev, NCH*M*sizeof(hipComplex)));

    // Allocate vector yMF in device
    CHECK (hipMalloc ((void**)&yMF, NCH*N*sizeof(hipComplex)));

    // Allocate matrix G in device
    CHECK (hipMalloc ((void**)&G, NCH*N*N*sizeof(hipComplex)));

    // Allocate vector x in device
    CHECK (hipMalloc ((void**)&x, NCH*N*sizeof(hipComplex)));

    // Allocate vector p in device
    CHECK (hipMalloc ((void**)&p, NCH*N*sizeof(hipComplex)));

    // Allocate vector Ap in device
    CHECK (hipMalloc ((void**)&e, N*sizeof(hipComplex)));

    // Allocate vector r in device
    CHECK (hipMalloc ((void**)&r, NCH*N*sizeof(hipComplex)));
    
    // Allocate vector Ar in device
    CHECK (hipMalloc ((void**)&m, N*sizeof(hipComplex)));

    // CHECK_CUBLAS (cublasSetMatrix (M, N, sizeof(*H), H, M, H_dev, M));

    // CHECK_CUBLAS (cublasSetMatrix (N, M, sizeof(*H), HH, N, HH_dev, N));

    // cublasSetVector() copies a vector x on the host to a vector on the GPU
    // CHECK_CUBLAS (cublasSetVector(M, sizeof (*y), y, 1, y_dev, 1)); //cp y->d_y

    /*** PREPROCESSING ***/
    /** Compute Gramian G = H^H*H **/
    // CHECK_CUBLAS (cublasCgemm (handle, CUBLAS_OP_N, CUBLAS_OP_N, N, N, M,
    //                         &al, HH_dev, N, H_dev, M,
    //                         &bet, G, N));
    createA(A);
    CHECK (hipMemcpy(G, A, N*SZ*sizeof(hipComplex), hipMemcpyHostToDevice));

    // /** Regularize A --> G + s^2*I **/
    // float s2 = 2.0f;
    // ScaleA <<< grid, block >>> (G, s2);
    // cudaDeviceSynchronize();

    /** Compute ymf = HH*y **/
    // CHECK_CUBLAS (cublasCgemv (handle, CUBLAS_OP_N, N, M, &al,
    //                             HH_dev, N, y_dev, 1, &bet, yMF, 1));
    createYMF(yMF_H);
    CHECK (hipMemcpy(yMF, yMF_H, SZ*sizeof(hipComplex), hipMemcpyHostToDevice));

    // Copy yMF -> p
    CHECK_CUBLAS (hipblasCcopy (handle, N, yMF, 1, p, 1));

    // Copy yMF -> r
    CHECK_CUBLAS (hipblasCcopy (handle, N, yMF, 1, r, 1));

    /***---CONJUGATE RESIDUAL METHOD---***/

    /** Compute Ap0 = Ar0 = e0 = m0 **/
    CHECK_CUBLAS (hipblasCgemv (handle, HIPBLAS_OP_N, N, N,&al,
                                G, N, yMF, 1, &bet, m, 1));
    // Copy m -> e
    CHECK_CUBLAS (hipblasCcopy (handle, N, m, 1, e, 1));

    for (int i = 0; i < 3; i++)
    {
        // Compute r^H dot m
        CHECK_CUBLAS (hipblasCdotc (handle, N, r, 1, m, 1, &alpha_divident));

        // Compute ||e||2
        CHECK_CUBLAS (hipblasScnrm2(handle, N, e, 1, &alpha_divisor));


        /** Calculate a **/
        alpha.x = alpha_divident.x / alpha_divisor;
        alpha.y = alpha_divident.y / alpha_divisor;

        // /** Compute x **/
        CHECK_CUBLAS ( hipblasCaxpy (handle, N, &alpha, p, 1, x, 1));
        
        // /** Compute r **/
        alpha.x = -alpha.x;
        alpha.y = -alpha.y;
        CHECK_CUBLAS (hipblasCaxpy (handle, N, &alpha, e, 1, r, 1));

        /** Compute Ar **/
        CHECK_CUBLAS (hipblasCgemv (handle, HIPBLAS_OP_N, N, N, &al,
                                    G, N, r, 1, &bet, m, 1));

        // Compute r^H dot m
        CHECK_CUBLAS (hipblasCdotc (handle, N, r, 1, m, 1, &beta_divident));

        /** Calculate b **/
        beta_divisor = alpha_divident;
        beta = hipCdivf(beta_divident, beta_divisor);

        // Compute p
        CHECK_CUBLAS (hipblasCaxpy (handle, N, &beta, p, 1, r, 1));

        // // Compute e
        CHECK_CUBLAS (hipblasCaxpy (handle, N, &alpha, e, 1, m, 1));
    }

    CHECK_CUBLAS (hipblasGetMatrix (N, N, sizeof(*A), G, N, A, N));

    CHECK_CUBLAS (hipblasGetVector (N, sizeof(*x), x, 1, x_H, 1));

    printf ("\nDetected Vector: \n");
    for (int j = 0; j < N; j++) {
        printf ("%.2f %.1f ", x_H[SZ - j - 1].x, x_H[SZ - j - 1].y);
    printf ("\n");
    }

    // destroy CUBLAS context
    hipblasDestroy(handle);

    // free device memory
    hipFree(H_dev);
    hipFree(G); 
    hipFree(yMF); 
    hipFree(x); 
    hipFree(p); 
    hipFree(r); 
    hipFree(e);
    hipFree(m);

    // free host memory
    free(x_H);
    free(H);
    free(HH);
    free(y);
    free(yMF_H);
    free(A);

    return EXIT_SUCCESS;
}

// nvcc -arch sm_50 CR.cu -lcublas -o a
// nvprof ./a
// nvprof --print-gpu-trace a