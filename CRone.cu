#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
// #include "cublas_v2.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

#define M 128           // Number of BS antennas
#define N 32            // Number of users
#define SC  1          // Number of subcarriers
#define SYM 1           // Number of symbols
#define NCH SC*SYM      // Number of channel matrices
#define SZ  SC*SYM*N    // Total threads

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(-10*error); \
    } \
}

void createA(hipComplex *A)
{
    for (int i = 0; i < N*SZ; i++)
    {
        A[i].x = 0.123f;    
        A[i].y = 0.123f;     
    }
}

void createYMF(hipComplex *y)
{
    for (int i = 0; i < SZ; i++)
    {
        y[i].x = 0.123f;    
        y[i].y = 0.456f;     
    }
}

__global__ void matVec(hipComplex* A, hipComplex* r, hipComplex* m) 
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < SZ) {

		hipComplex tmp = {0.0f, 0.0f};

		for (int i = 0; i < N; i++) {
            tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], A[idx*N*N + i]));
		}
		m[idx*N] = tmp;
	}
}

__global__ void Caxpy(hipComplex alpha, hipComplex *p, hipComplex *r) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < SZ)
    {
        p[idx*N] = hipCaddf(r[idx*N], hipCmulf(alpha, p[idx*N]));
    }
}

__global__ void Cymax(hipComplex beta, hipComplex *p, hipComplex *r) 
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < SZ)
    {
        p[idx*N] = hipCsubf(r[idx*N], hipCmulf(beta, p[idx*N]));
    }
}


__global__ void Dot(hipComplex *r, hipComplex *m)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hipComplex tmp = {0.0f, 0.0f};

    for (int i = 0; i < N; i++)
        tmp = hipCaddf(tmp, hipCmulf(hipConjf(r[idx*N + i]), m[idx*N + i]));
}

__global__ void Norm(hipComplex *e)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    float tmpf = 0.0f;
    for (int i = 0; i < N; i++)
        tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
    tmpf = sqrtf(tmpf);
}

__global__ void CR (hipComplex *A, hipComplex *r, hipComplex *p, hipComplex *m, hipComplex *e, hipComplex *x)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < SZ) {

        float tmpf = 0.0f;
		hipComplex tmp = {0.0f, 0.0f};
        hipComplex alpha = {0.0f, 0.0f};
        hipComplex beta = {0.0f, 0.0f};
        hipComplex alpha_beta = {0.0f, 0.0f};

		for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++){
                tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], A[idx*N*N + i*N + j]));
            }

            m[idx*N + i] = tmp;
            e[idx*N + i] = tmp;
            // alpha divident
            alpha = hipCaddf(alpha, hipCmulf(hipConjf(r[idx*N + i]), tmp));
            tmpf += (tmp.x * tmp.x) + (tmp.y * tmp.y);
		}
        // alpha divisor
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;

        // alpha
        alpha.x = alpha.x / tmpf;
        alpha.y = alpha.y / tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCmulf(alpha, p[idx*N + i]);
            r[idx*N + i] = hipCsubf(r[idx*N + i], hipCmulf(alpha, e[idx*N + i]));
        }

        beta = {0.0f, 0.0f};
		for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++)
                tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], A[idx*N*N + i*N + j]));

            m[idx*N + i] = tmp;
            // beta divident
            beta = hipCaddf(beta, hipCmulf(hipConjf(r[idx*N + i]), tmp));
		}

        alpha = beta;
        beta = hipCdivf(beta, alpha_beta);

        tmpf = 0.0f;
        for (int i = 0; i < N; i++)
        {
            p[idx*N + i] = hipCaddf(r[idx*N + i], hipCmulf(beta, p[idx*N + i]));
            e[idx*N + i] = hipCaddf(m[idx*N + i], hipCmulf(beta, e[idx*N + i]));
            tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
        }
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;
        alpha.x = alpha.x/tmpf;
        alpha.y = alpha.y/tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCaddf(x[idx*N + i], hipCmulf(alpha, p[idx*N + i]));
            r[idx*N + i] = hipCsubf(r[idx*N + i], hipCmulf(alpha, e[idx*N + i]));
        }

        beta = {0.0f, 0.0f};
        for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++)
                tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], A[idx*N*N + i*N + j]));

            m[idx*N + i] = tmp;
            beta = hipCaddf(beta, hipCmulf(hipConjf(r[idx*N + i]), tmp));
		}
        alpha = beta;
        beta = hipCdivf(beta, alpha_beta);

        tmpf = 0.0f;
        for (int i = 0; i < N; i++)
        {
            p[idx*N + i] = hipCaddf(r[idx*N + i], hipCmulf(beta, p[idx*N + i]));
            e[idx*N + i] = hipCaddf(m[idx*N + i], hipCmulf(beta, e[idx*N + i]));
            tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
        }
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;
        alpha.x = alpha.x/tmpf;
        alpha.y = alpha.y/tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCaddf(x[idx*N + i], hipCmulf(alpha, p[idx*N + i]));
        }
	}
}

__global__ void CRVec (hipComplex *A, hipComplex *r, hipComplex *p, hipComplex *m, hipComplex *e, hipComplex *x)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ hipComplex r_s[SC*N];
    if(idx < SZ)
    {
        for (int i = 0; i < N; i++)
            r_s[threadIdx.x + i] = r[idx*N + i];
    }

	if (idx < SZ) {

        float tmpf = 0.0f;
		hipComplex tmp = {0.0f, 0.0f};
        hipComplex alpha = {0.0f, 0.0f};
        hipComplex beta = {0.0f, 0.0f};
        hipComplex alpha_beta = {0.0f, 0.0f};

		for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++){
                tmp = hipCaddf(tmp, hipCmulf(r_s[threadIdx.x + i], A[idx*N*N + i*N + j]));
                // printf("Tid %d R %f A%f\n", idx, r[idx*N + i].x, A[idx*N*N + i*N + j].y);
            }

            m[idx*N + i] = tmp;
            e[idx*N + i] = tmp;
            // alpha divident
            alpha = hipCaddf(alpha, hipCmulf(hipConjf(r_s[threadIdx.x + i]), tmp));
            tmpf += (tmp.x * tmp.x) + (tmp.y * tmp.y);
		}
        // alpha divisor
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;

        // alpha
        alpha.x = alpha.x / tmpf;
        alpha.y = alpha.y / tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCmulf(alpha, p[idx*N + i]);
            r_s[threadIdx.x + i] = hipCsubf(r_s[threadIdx.x + i], hipCmulf(alpha, e[idx*N + i]));
        }

        beta = {0.0f, 0.0f};
		for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++)
                tmp = hipCaddf(tmp, hipCmulf(r_s[threadIdx.x + i], A[idx*N*N + i*N + j]));

            m[idx*N + i] = tmp;
            // beta divident
            beta = hipCaddf(beta, hipCmulf(hipConjf(r_s[threadIdx.x + i]), tmp));
		}

        alpha = beta;
        beta = hipCdivf(beta, alpha_beta);

        tmpf = 0.0f;
        for (int i = 0; i < N; i++)
        {
            p[idx*N + i] = hipCaddf(r_s[threadIdx.x + i], hipCmulf(beta, p[idx*N + i]));
            e[idx*N + i] = hipCaddf(m[idx*N + i], hipCmulf(beta, e[idx*N + i]));
            tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
        }
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;
        alpha.x = alpha.x/tmpf;
        alpha.y = alpha.y/tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCaddf(x[idx*N + i], hipCmulf(alpha, p[idx*N + i]));
            r_s[threadIdx.x + i] = hipCsubf(r_s[threadIdx.x + i], hipCmulf(alpha, e[idx*N + i]));
        }

        beta = {0.0f, 0.0f};
        for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++)
                tmp = hipCaddf(tmp, hipCmulf(r_s[threadIdx.x + i], A[idx*N*N + i*N + j]));

            m[idx*N + i] = tmp;
            beta = hipCaddf(beta, hipCmulf(hipConjf(r_s[threadIdx.x + i]), tmp));
		}
        alpha = beta;
        beta = hipCdivf(beta, alpha_beta);

        tmpf = 0.0f;
        for (int i = 0; i < N; i++)
        {
            p[idx*N + i] = hipCaddf(r_s[threadIdx.x + i], hipCmulf(beta, p[idx*N + i]));
            e[idx*N + i] = hipCaddf(m[idx*N + i], hipCmulf(beta, e[idx*N + i]));
            tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
        }
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;
        alpha.x = alpha.x/tmpf;
        alpha.y = alpha.y/tmpf;

        for (int i = 0; i < N; i++)
            x[idx*N + i] = hipCaddf(x[idx*N + i], hipCmulf(alpha, p[idx*N + i]));
	}
}

__global__ void CRMat (hipComplex *A, hipComplex *r, hipComplex *p, hipComplex *m, hipComplex *e, hipComplex *x)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ hipComplex G[SC*N*N];

    if(idx < SZ)
    {
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++){
                G[threadIdx.x*N + i*N + j] = A[idx*N*N + i*N + j];
            }
    }

	if (idx < SZ) 
    {

        float tmpf = 0.0f;
		hipComplex tmp = {0.0f, 0.0f};
        hipComplex alpha = {0.0f, 0.0f};
        hipComplex beta = {0.0f, 0.0f};
        hipComplex alpha_beta = {0.0f, 0.0f};

		for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++){
                tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], G[threadIdx.x + i*N + j]));
            }

            m[idx*N + i] = tmp;
            e[idx*N + i] = tmp;
            // alpha divident
            alpha = hipCaddf(alpha, hipCmulf(hipConjf(r[idx*N + i]), tmp));
            tmpf += (tmp.x * tmp.x) + (tmp.y * tmp.y);
		}
        // alpha divisor
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;

        // alpha
        alpha.x = alpha.x / tmpf;
        alpha.y = alpha.y / tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCmulf(alpha, p[idx*N + i]);
            r[idx*N + i] = hipCsubf(r[idx*N + i], hipCmulf(alpha, e[idx*N + i]));
        }

        beta = {0.0f, 0.0f};
		for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++)
                tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], G[threadIdx.x + i*N + j]));

            m[idx*N + i] = tmp;
            // beta divident
            beta = hipCaddf(beta, hipCmulf(hipConjf(r[idx*N + i]), tmp));
		}

        alpha = beta;
        beta = hipCdivf(beta, alpha_beta);

        tmpf = 0.0f;
        for (int i = 0; i < N; i++)
        {
            p[idx*N + i] = hipCaddf(r[idx*N + i], hipCmulf(beta, p[idx*N + i]));
            e[idx*N + i] = hipCaddf(m[idx*N + i], hipCmulf(beta, e[idx*N + i]));
            tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
        }
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;
        alpha.x = alpha.x/tmpf;
        alpha.y = alpha.y/tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCaddf(x[idx*N + i], hipCmulf(alpha, p[idx*N + i]));
            r[idx*N + i] = hipCsubf(r[idx*N + i], hipCmulf(alpha, e[idx*N + i]));
        }

        beta = {0.0f, 0.0f};
        for (int i = 0; i < N; i++) {
            tmp = {0.0f, 0.0f};
            for (int j = 0; j < N; j++)
                tmp = hipCaddf(tmp, hipCmulf(r[idx*N + i], G[threadIdx.x + i*N + j]));

            m[idx*N + i] = tmp;
            beta = hipCaddf(beta, hipCmulf(hipConjf(r[idx*N + i]), tmp));
		}
        alpha = beta;
        beta = hipCdivf(beta, alpha_beta);

        tmpf = 0.0f;
        for (int i = 0; i < N; i++)
        {
            p[idx*N + i] = hipCaddf(r[idx*N + i], hipCmulf(beta, p[idx*N + i]));
            e[idx*N + i] = hipCaddf(m[idx*N + i], hipCmulf(beta, e[idx*N + i]));
            tmpf += (e[idx*N + i].x * e[idx*N + i].x) + (e[idx*N + i].y * e[idx*N + i].y);
        }
        tmpf = sqrtf(tmpf);
        alpha_beta = alpha;
        alpha.x = alpha.x/tmpf;
        alpha.y = alpha.y/tmpf;

        for (int i = 0; i < N; i++)
        {
            x[idx*N + i] = hipCaddf(x[idx*N + i], hipCmulf(alpha, p[idx*N + i]));
        }
	}
}

int main (void)
{
    // Get device information
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK (hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);

    // Set up device
	CHECK (hipSetDevice(dev));

    hipComplex * H;          // Chanel matrix in host
    hipComplex * HH;         // Hermitian Chanel matrix in host

    hipComplex * H_dev;      // Chanel matrix in device
    hipComplex * HH_dev;     // Hermitian Chanel matrix in device

    hipComplex * G;          // Gram matrix in device

    hipComplex * y;          // Received vector in host
    hipComplex * y_dev;      // Received vector in device

    hipComplex * yMF_H;      // Matched filter in host
    hipComplex * yMF;        // Matched filter in device

    hipComplex * A;          // Matrix A in host

    hipComplex * x_H;        // Detected vector 

    // Five vectors of storage are needed in addition to the matrix A: x, p, Ap, r, Ar
    hipComplex * x;
    hipComplex * p;
    hipComplex * e; // Ap
    hipComplex * r;
    hipComplex * m; // Ar

    dim3 block (SC);
    dim3 grid  (SYM);

    // Matrix H in host
    H = (hipComplex*) malloc (NCH * M * N * sizeof(hipComplex));
    if (!H) {
        printf ("Host memory allocation failed for H");
        return EXIT_FAILURE;
    }

    // Matrix H in host
    HH = (hipComplex*)malloc (NCH * M * N * sizeof(hipComplex));
    if (!HH) {
        printf ("Host memory allocation failed for HH");
        return EXIT_FAILURE;
    }

    // Received vector y in host
    y = (hipComplex*)malloc (NCH * M * sizeof(hipComplex));
    if (!y) {
        printf ("Host memory allocation failed for y");
        return EXIT_FAILURE;
    }

    // Detected vector y in host
    yMF_H = (hipComplex*)malloc (NCH * N * sizeof(hipComplex));
    if (!yMF_H) {
        printf ("Host memory allocation failed for yMF_H");
        return EXIT_FAILURE;
    }

    // Detected vector y in host
    x_H = (hipComplex*)malloc (NCH * N * sizeof(hipComplex));
    if (!x_H) {
        printf ("Host memory allocation failed for x_H");
        return EXIT_FAILURE;
    }

    // Matrix H in host
    A = (hipComplex*)malloc (NCH * N * N * sizeof(hipComplex));
    if (!A) {
        printf ("Host memory allocation failed for A");
        return EXIT_FAILURE;
    }

    // Allocate matrix H in device
    CHECK (hipMalloc ((void**)&H_dev, NCH*M*N*sizeof(hipComplex)));

    // Allocate matrix HH in device
    CHECK (hipMalloc ((void**)&HH_dev, NCH*M*N*sizeof(hipComplex)));

    // Allocate vector y in device
    CHECK (hipMalloc ((void**)&y_dev, NCH*M*sizeof(hipComplex)));

    // Allocate vector yMF in device
    CHECK (hipMalloc ((void**)&yMF, NCH*N*sizeof(hipComplex)));

    // Allocate matrix G in device
    CHECK (hipMalloc ((void**)&G, NCH*N*N*sizeof(hipComplex)));

    // Allocate vector x in device
    CHECK (hipMalloc ((void**)&x, NCH*N*sizeof(hipComplex)));

    // Allocate vector p in device
    CHECK (hipMalloc ((void**)&p, NCH*N*sizeof(hipComplex)));

    // Allocate vector Ap in device
    CHECK (hipMalloc ((void**)&e, N*sizeof(hipComplex)));

    // Allocate vector r in device
    CHECK (hipMalloc ((void**)&r, NCH*N*sizeof(hipComplex)));
    
    // Allocate vector Ar in device
    CHECK (hipMalloc ((void**)&m, N*sizeof(hipComplex)));

    /*** PREPROCESSING ***/
    /** Compute Gramian G = H^H*H **/
    createA(A);
    CHECK (hipMemcpy(G, A, N*SZ*sizeof(hipComplex), hipMemcpyHostToDevice));

    // /** Regularize A --> G + s^2*I **/

    /** Compute ymf = HH*y **/
    createYMF(yMF_H);
    CHECK (hipMemcpy(r, yMF_H, SZ*sizeof(hipComplex), hipMemcpyHostToDevice));
    CHECK (hipMemcpy(p, yMF_H, SZ*sizeof(hipComplex), hipMemcpyHostToDevice));

    /***---CONJUGATE RESIDUAL METHOD---***/
    CR <<< grid, block >>> (G, r, p, m, e, x);
    CHECK (hipMemcpy(x_H, x, SZ*sizeof(hipComplex), hipMemcpyDeviceToHost));

    printf ("\nDetected Vector: \n");
    for (int j = 0; j < N; j++) {
        printf ("%.2f %.1f ", x_H[SZ - j-1].x, x_H[SZ - j-1].y);
    printf ("\n");
    }

    // free device memory
    hipFree(H_dev);
    hipFree(G); 
    hipFree(yMF); 
    hipFree(x); 
    hipFree(p); 
    hipFree(r); 
    hipFree(e);
    hipFree(m);

    // free host memory
    free(H);
    free(HH);
    free(y);
    free(yMF_H);
    free(A);

    return EXIT_SUCCESS;
}